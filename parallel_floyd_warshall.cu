#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <limits.h>

#include "parallel_floyd_warshall.cuh"

#define min(a,b) (((a)<(b))?(a):(b))
#define BLOCK_SIZE 16

static size_t data_size = 10;

//state variable
static uint64_t rand_state;

__global__ void parallel_floyd_warshall(int *graph, int n, int *path)
{
   // block indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = n * BLOCK_SIZE * by;
    int aEnd = aBegin + n - 1;
    int aStep = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;
    int bStep = BLOCK_SIZE * n;
    
    int pathsub = 0;

    for(int a = aBegin, b = bBegin;  a <= aEnd; a += aStep, b += bStep) {
	    //load block into shared memory
	    __shared__ int graph_s[BLOCK_SIZE][BLOCK_SIZE];
	    __shared__ int path_s[BLOCK_SIZE][BLOCK_SIZE];
      	    graph_s[ty][tx] = graph[a + n * ty + tx];
	    path_s[ty][tx] = path[b + n * ty + tx];
	    __syncthreads();
	    
	    //find minimum for block
	    for(int k = 0; k < BLOCK_SIZE; ++k) {
		    pathsub = graph_s[ty][k] < graph_s[ty][k] + path_s[k][tx] ?
			    graph_s[ty][k] : graph_s[ty][k] + path_s[k][tx];
	    }
	    __syncthreads();		   
    }   
    //writeback
    int pathwrite =  n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    path[pathwrite + n * ty + tx] = pathsub;

}

int  *serial_floyd_warshall(int *graph, int n)
{
	int *path = (int *) calloc(sizeof(int), n*n);
	memcpy(graph, path, n*n);
	
	for(int k=0; k < n; k++) {
		for(int i=0; i < n; i++) {
			for(int j=0; j < n; j++) {
				path[(i * n) + j] = min(path[(i * n) + j], path[(i * n) + k]+path[(k * n) + j]);
			}
		}
	}
	return path;
}

struct timespec timer_start()
{
	struct timespec start_time;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start_time);
	return start_time;
}

long int timer_end(struct timespec start_time)
{
	struct timespec end_time;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &end_time);
	long int diff = (end_time.tv_sec - start_time.tv_sec) *
		(long)1e9 + (end_time.tv_nsec - start_time.tv_nsec);
	return diff;
}

uint64_t xorshift(void)
{
	uint64_t u = rand_state;
	u ^= u << 12;
	u ^= u << 25;
	u ^= u >> 27;
	rand_state = u;
	return u * 0x2545F4914F6CDD1D;
}

uint64_t xrand(void)
{
	return xorshift(); 
}

void xseed(uint64_t seed)
{
	rand_state = seed;
}

int main(int argc, char ** argv)
{
	//seed RNG
	xseed(time(NULL));
	
	unsigned long int n = 0;
	if(argc != 2) {
		n = data_size;
	}
	else {
		n = atoi(argv[1]);
	}
	
	//create graph
	int *graph = (int *) calloc(sizeof(int), n * n);
	for(int i=0; i < n; i++) {
		for(int j=0; j < n; j++) {
			if(i == j){
				graph[(i * n) + j] = 0;
			}
			else {
				graph[(i * n) + j] = xrand();
			}
		}
	}

	int *path = NULL;
	
	//test serial code speed
	struct timespec start = timer_start();
	path = serial_floyd_warshall(graph, n);
	printf("serial Floyd-Warshall: %li nanoseconds\n", timer_end(start));   

	int *graph_d;
	hipMalloc(&graph_d, n*n);
	hipMemcpy(graph_d, graph, n*n, hipMemcpyHostToDevice);

	int *path_d;
	hipMalloc(&path_d, n*n);

	int grid = 1;
	int block = 1;
	
	//test parallel code speed
	start = timer_start();
	parallel_floyd_warshall<<<grid, block>>>(graph_d, n, path_d);
	printf("parallel Floyd-Warshall: %li nanoseconds\n", timer_end(start));


	//free memory
	hipFree(path_d);
	hipFree(graph_d);
	free(path);
	free(graph);
	return 0;
}
